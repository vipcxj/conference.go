#include "hip/hip_runtime.h"
#include "gst-cu.cuh"
#include <hip/hip_fp16.h>

__global__ void _copy_ai_input(unsigned char * d_src, int stride, half * d_target, int frame)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < 224 && row < 224)
    {
        half * half_target = (half *) d_target;
        half_target[0 * 16 * 224 * 224 + frame * 224 * 224 + row * 224 + col] = __hmul(__hadd(__hdiv(__int2half_rn(d_src[row * stride + col * 3]), 256), __float2half(-0.485f)), 4.36681f);
        half_target[1 * 16 * 224 * 224 + frame * 224 * 224 + row * 224 + col] = __hmul(__hadd(__hdiv(__int2half_rn(d_src[row * stride + col * 3 + 1]), 256), __float2half(-0.456f)), 4.464286f);
        half_target[2 * 16 * 224 * 224 + frame * 224 * 224 + row * 224 + col] = __hmul(__hadd(__hdiv(__int2half_rn(d_src[row * stride + col * 3 + 2]), 256), __float2half(-0.406f)), 4.444444f);
    }
}

namespace cfgo
{
    namespace gst
    {
        void copy_ai_input(unsigned char * d_src, int stride, half * d_target, int frame, hipStream_t stream)
        {
            _copy_ai_input<<<{4, 4}, {8, 8}, 0, stream>>>(d_src, stride, (half *) d_target, frame);
        }
    } // namespace gst
    
} // namespace cfgo
