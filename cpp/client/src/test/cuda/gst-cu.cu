#include "hip/hip_runtime.h"
#include "gst-cu.cuh"
#include <hip/hip_fp16.h>
#include <stdio.h>

template<unsigned int channel>
__device__ inline void _copy_frame_(unsigned int width, unsigned int height, unsigned int channels, unsigned int frames, unsigned char * d_src, int stride, unsigned char * d_target, int frame, int index, int col, int row)
{
    _copy_frame_<channel - 1>(width, height, channels, frames, d_src, stride, d_target, frame, index, col, row);
    auto v = d_src[row * stride + col * channels + channel];
    d_target[index * channels * frames * width * height + channel * frames * width * height + frame * width * height + row * width + col] = v;
}

template<>
__device__ inline void _copy_frame_<0U>(unsigned int width, unsigned int height, unsigned int channels, unsigned int frames, unsigned char * d_src, int stride, unsigned char * d_target, int frame, int index, int col, int row)
{
    auto v = d_src[row * stride + col * channels];
    d_target[index * channels * frames * width * height + frame * width * height + row * width + col] = v;
}

template<unsigned int width, unsigned int height, unsigned int channels, unsigned int frames>
__global__ void _copy_frame(unsigned char * d_src, int stride, unsigned char * d_target, int frame, int index)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < width && row < height)
    {   
        _copy_frame_<channels - 1>(width, height, channels, frames, d_src, stride, d_target, frame, index, col, row);
    }
}

template<unsigned int frame>
__device__ inline void _copy_ai_input_(unsigned int width, unsigned int height, unsigned int frames, unsigned char * d_src, int src_index, half * d_target, int tgt_index, int col, int row)
{
    _copy_ai_input_<frame - 1>(width, height, frames, d_src, src_index, d_target, tgt_index, col, row);
    half h1 = __hmul(__hadd(__hdiv(__int2half_rn(d_src[src_index * 3 * frames * width * height + 0 * frames * width * height + frame * width * height + row * width + col]), 256), __float2half(-0.485f)), 4.36681f);
    d_target[tgt_index * 3 * frames * width * height + 0 * frames * width * height + frame * width * height + row * width + col] = h1;
    half h2 = __hmul(__hadd(__hdiv(__int2half_rn(d_src[src_index * 3 * frames * width * height + 1 * frames * width * height + frame * width * height + row * width + col]), 256), __float2half(-0.456f)), 4.464286f);
    d_target[tgt_index * 3 * frames * width * height + 1 * frames * width * height + frame * width * height + row * width + col] = h2;
    half h3 = __hmul(__hadd(__hdiv(__int2half_rn(d_src[src_index * 3 * frames * width * height + 2 * frames * width * height + frame * width * height + row * width + col]), 256), __float2half(-0.406f)), 4.444444f);
    d_target[tgt_index * 3 * frames * width * height + 2 * frames * width * height + frame * width * height + row * width + col] = h3;
}

template<>
__device__ inline void _copy_ai_input_<0U>(unsigned int width, unsigned int height, unsigned int frames, unsigned char * d_src, int src_index, half * d_target, int tgt_index, int col, int row)
{
    half h1 = __hmul(__hadd(__hdiv(__int2half_rn(d_src[src_index * 3 * frames * width * height + 0 * frames * width * height + row * width + col]), 256), __float2half(-0.485f)), 4.36681f);
    d_target[tgt_index * 3 * frames * width * height + 0 * frames * width * height + row * width + col] = h1;
    half h2 = __hmul(__hadd(__hdiv(__int2half_rn(d_src[src_index * 3 * frames * width * height + 1 * frames * width * height + row * width + col]), 256), __float2half(-0.456f)), 4.464286f);
    d_target[tgt_index * 3 * frames * width * height + 1 * frames * width * height + row * width + col] = h2;
    half h3 = __hmul(__hadd(__hdiv(__int2half_rn(d_src[src_index * 3 * frames * width * height + 2 * frames * width * height + row * width + col]), 256), __float2half(-0.406f)), 4.444444f);
    d_target[tgt_index * 3 * frames * width * height + 2 * frames * width * height + row * width + col] = h3;
}

template<unsigned int width, unsigned int height, unsigned int frames>
__global__ void _copy_ai_input(unsigned char * d_src, int src_index, half * d_target, int tgt_index)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < width && row < height)
    {
        _copy_ai_input_<frames - 1>(width, height, frames, d_src, src_index, d_target, tgt_index, col, row);
    }
}

namespace cfgo
{
    namespace gst
    {
        void copy_frame(unsigned char * d_src, int stride, unsigned char * d_target, int frame, int index, hipStream_t stream)
        {
            _copy_frame<224, 224, 3, 16><<<{16, 16}, {16, 16}, 0, stream>>>(d_src, stride , d_target, frame, index);
            cudaCheckErrors("copy_frame");
        }
        void copy_ai_input(unsigned char * d_src, int src_index, half * d_target, int tgt_index, hipStream_t stream)
        {
            _copy_ai_input<224, 224, 16><<<{16, 16}, {16, 16}, 0, stream>>>(d_src, src_index, d_target, tgt_index);
            cudaCheckErrors("copy_ai");
        }
    } // namespace gst
    
} // namespace cfgo
